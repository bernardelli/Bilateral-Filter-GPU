#include "hip/hip_runtime.h"
/*
Slicing:

Perform slicing and nonlinearity. Uses texture unities.
*/

#include "slicing.cuh"

texture<float, 3> wi_tex;
texture<float, 3> w_tex;

__global__ void slicing( float *dev_image, const dim3 imsize, int scale_xy, int scale_eps)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < imsize.x) && (j < imsize.y))
	{

		float k = dev_image[i + imsize.x*j];

		float x = 0.5f + (float)i / (float)scale_xy;
		float y = 0.5f + (float)j / (float)scale_xy;
		float z = 0.5f + (float)k / (float)scale_eps;

		dev_image[i + imsize.x*j] = tex3D(wi_tex, x, y, z) / tex3D(w_tex, x, y, z);
		
	}

}



float callingSlicing(float* dev_image, const float *dev_cube_wi, const float *dev_cube_w, const dim3 imsize, int scale_xy, int scale_eps, dim3 dimensions_down)
{


	int slicing_status = 0;
	hipError_t cudaStatus;


	/****************************************************************************************************************
	***	Set Texture proprierties																				*****
	*****************************************************************************************************************/
	wi_tex.filterMode = hipFilterModeLinear;
	wi_tex.addressMode[0] = hipAddressModeClamp; 
	wi_tex.addressMode[1] = hipAddressModeClamp;
	wi_tex.addressMode[2] = hipAddressModeClamp;
	w_tex.filterMode = hipFilterModeLinear;      
	w_tex.addressMode[0] = hipAddressModeClamp;
	w_tex.addressMode[1] = hipAddressModeClamp;
	w_tex.addressMode[2] = hipAddressModeClamp;

	/****************************************************************************************************************
	***	Create 3D Arrays																						*****
	*****************************************************************************************************************/

	hipArray *dev_cube_wi_array, *dev_cube_w_array;
	hipExtent extent = make_hipExtent( dimensions_down.x, dimensions_down.y, dimensions_down.z); 
	hipChannelFormatDesc channelFloat = hipCreateChannelDesc<float>();

	cudaStatus = hipMalloc3DArray(&dev_cube_wi_array, &channelFloat, extent);
	cudaStatus = hipMalloc3DArray(&dev_cube_w_array, &channelFloat, extent);

	if (cudaStatus != hipSuccess) {
		std::cout << "error on malloc3darray " << hipGetErrorString(cudaStatus) << std::endl;
	}

	/*Copy arrays*/

	hipMemcpy3DParms copyParams1 = { 0 };
	copyParams1.srcPtr = make_hipPitchedPtr((void*) dev_cube_wi,
		dimensions_down.x* sizeof(float), //https://devtalk.nvidia.com/default/topic/481806/copy-3d-data-from-host-to-device/
		dimensions_down.x, dimensions_down.y);
	copyParams1.dstArray = dev_cube_wi_array;
	copyParams1.extent = extent;
	copyParams1.kind = hipMemcpyDeviceToDevice;
	cudaStatus = hipMemcpy3D(&copyParams1);

	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array1" << std::endl;
		slicing_status = 1;
	}
	hipMemcpy3DParms copyParams2 = { 0 };
	copyParams2.srcPtr = make_hipPitchedPtr((void*) dev_cube_w,
		dimensions_down.x * sizeof(float),
		dimensions_down.x, dimensions_down.y);
	copyParams2.dstArray = dev_cube_w_array;
	copyParams2.extent = extent;
	copyParams2.kind = hipMemcpyDeviceToDevice;
	cudaStatus = hipMemcpy3D(&copyParams2);
	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array2"<< std::endl;
		slicing_status = 1;
	}

	/*Get texture references*/
	const struct textureReference * wi_tex_ref;
	const struct textureReference * w_tex_ref;
	
#if CUDA_VERSION < 5000 /* 5.0 */
	cudaStatus = hipGetTextureReference(&wi_tex_ref, HIP_SYMBOL("wi_tex"));
	hipGetTextureReference(&w_tex_ref, HIP_SYMBOL("w_tex"));
#else
	cudaStatus = hipGetTextureReference(&wi_tex_ref, HIP_SYMBOL(&wi_tex));
	hipGetTextureReference(&w_tex_ref, HIP_SYMBOL(&w_tex));
#endif

	
	if (cudaStatus != hipSuccess) {
		std::cout << "error on gettexref " << hipGetErrorString(cudaStatus) << std::endl;
	}
	
	/*Bind textures to array*/
	cudaStatus = hipBindTextureToArray(wi_tex_ref, dev_cube_wi_array, &channelFloat);
	cudaStatus = hipBindTextureToArray(w_tex_ref, dev_cube_w_array, &channelFloat);
	
	if (cudaStatus != hipSuccess) {
		std::cout << "error on bind text " << hipGetErrorString(cudaStatus) << std::endl;
	}
	
	/****************************************************************************************************************
	***	Actual Slicing kernel																					*****
	*****************************************************************************************************************/

	const dim3 block(16, 16);
	const dim3 grid(((imsize.x + block.x - 1) / block.x), ((imsize.y + block.y - 1) / block.y));
	
	hipEvent_t start_1, stop_1;
    float time_1;
    hipEventCreate(&start_1);
    hipEventCreate(&stop_1);

    hipEventRecord(start_1);

	slicing <<< grid, block >>> (dev_image, imsize, scale_xy, scale_eps);
	
	hipEventRecord(stop_1);
    hipEventSynchronize(stop_1);

    hipEventElapsedTime(&time_1, start_1, stop_1);
	float time = time_1;

	/*clean*/
	hipUnbindTexture(wi_tex_ref);
	hipUnbindTexture(w_tex_ref);
	hipFreeArray(dev_cube_wi_array);
	hipFreeArray(dev_cube_w_array);
	return time;
}
