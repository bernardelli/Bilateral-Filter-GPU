#include "hip/hip_runtime.h"
#include "slicing.h"

texture<float, 3> wi_tex;
texture<float, 3> w_tex;

__global__ void slicing( float *dev_image, const dim3 imsize, int scale_xy, int scale_eps)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((i < imsize.x) && (j < imsize.y))
	{

		float k = dev_image[i + imsize.x*j];


		dev_image[i + imsize.x*j] = 256*tex3D(wi_tex, 0.5f + (float)i / (float)scale_xy, 0.5f + (float)j / (float)scale_xy, 0.5f + (float)k / (float)scale_eps) 
									/ tex3D(w_tex, 0.5f + (float)i / (float)scale_xy, 0.5f + (float)j / (float)scale_xy, 0.5f + (float)k / (float)scale_eps);
		
	}

}

/*__global__ void fill_arrays(hipArray* dev_cube_wi_array, hipArray* dev_cube_w_array, const float* dev_cube_wi, const float* dev_cube_w, const dim3 )
{

	const int ix = blockDim.x*blockIdx.x + threadIdx.x;
	const int iy = blockDim.y*blockIdx.y + threadIdx.y;
	const int iz = blockIdx.z;
	const int cube_idx = ix + iy*dimensions_down.x + iz*dimensions_down.x*dimensions_down.y;
	if(ix < dimensions_down.x &&  iy < dimensions_down.y && iy < dimensions_down.y )
	{
		dev_cube_wi_array = dev_cube_wi[cube_idx];
	}
}*/

float callingSlicing(float* dev_image, const float *dev_cube_wi, const float *dev_cube_w, const dim3 imsize, int scale_xy, int scale_eps, dim3 dimensions_down)
{
	int slicing_status = 0;
	const dim3 block2(16, 16);
	wi_tex.filterMode = hipFilterModeLinear;      // linear interpolation
	wi_tex.addressMode[0] = hipAddressModeClamp; //hipAddressModeClamp
	wi_tex.addressMode[1] = hipAddressModeClamp;
	wi_tex.addressMode[2] = hipAddressModeClamp;
	w_tex.filterMode = hipFilterModeLinear;      // linear interpolation
	w_tex.addressMode[0] = hipAddressModeClamp;
	w_tex.addressMode[1] = hipAddressModeClamp;
	w_tex.addressMode[2] = hipAddressModeClamp;
	hipExtent extent1 = make_hipExtent( dimensions_down.x*sizeof(float), dimensions_down.y, dimensions_down.z); 
	hipExtent extent2 = make_hipExtent( dimensions_down.x, dimensions_down.y, dimensions_down.z); 
	hipArray *dev_cube_wi_array, *dev_cube_w_array;

	hipChannelFormatDesc channelFloat = hipCreateChannelDesc<float>();
	hipError_t cudaStatus = hipMalloc3DArray(&dev_cube_wi_array, &channelFloat, extent2);
	cudaStatus = hipMalloc3DArray(&dev_cube_w_array, &channelFloat, extent2);
	if (cudaStatus != hipSuccess) {
		std::cout << "error on malloc3darray " << hipGetErrorString(cudaStatus) << std::endl;
	}

	hipMemcpy3DParms copyParams1 = { 0 };
	copyParams1.srcPtr = make_hipPitchedPtr((void*) dev_cube_wi,
		dimensions_down.x* sizeof(float), //https://devtalk.nvidia.com/default/topic/481806/copy-3d-data-from-host-to-device/
		dimensions_down.x, dimensions_down.y);
	copyParams1.dstArray = dev_cube_wi_array;
	copyParams1.extent = extent2;
	copyParams1.kind = hipMemcpyDeviceToDevice;
	cudaStatus = hipMemcpy3D(&copyParams1);

	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array1" << std::endl;
		slicing_status = 1;
	}
	hipMemcpy3DParms copyParams2 = { 0 };
	copyParams2.srcPtr = make_hipPitchedPtr((void*) dev_cube_w,
		dimensions_down.x * sizeof(float),
		dimensions_down.x, dimensions_down.y);
	copyParams2.dstArray = dev_cube_w_array;
	copyParams2.extent = extent2;
	copyParams2.kind = hipMemcpyDeviceToDevice;
	cudaStatus = hipMemcpy3D(&copyParams2);
	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array2"<< std::endl;
		slicing_status = 1;
	}
	//hipMemcpyToArray(dev_cube_wi_array, 0, 0, dev_cube_wi, dimensions_down.x*dimensions_down.y*dimensions_down.z, hipMemcpyDeviceToDevice);
	//hipMemcpyToArray(dev_cube_w_array, 0, 0, dev_cube_w, dimensions_down.x*dimensions_down.y*dimensions_down.z, hipMemcpyDeviceToDevice);
	//fill_arrays<<grid,block>>(dev_cube_wi_array, dev_cube_w_array,dev_cube_wi, dev_cube_w,dimensions_down);

	//struct hipChannelFormatDesc descr = hipCreateChannelDesc((int)dimensions_down.x, (int)dimensions_down.y, (int)dimensions_down.z, hipChannelFormatKindFloat);
	const struct textureReference * wi_tex_ref;
	const struct textureReference * w_tex_ref;
	
#if CUDA_VERSION < 5000 /* 5.0 */
	cudaStatus = hipGetTextureReference(&wi_tex_ref, HIP_SYMBOL("wi_tex"));
	hipGetTextureReference(&w_tex_ref, HIP_SYMBOL("w_tex"));
#else
	cudaStatus = hipGetTextureReference(&wi_tex_ref, HIP_SYMBOL(&wi_tex));
	hipGetTextureReference(&w_tex_ref, HIP_SYMBOL(&w_tex));
#endif

	
	if (cudaStatus != hipSuccess) {
		std::cout << "error on gettexref " << hipGetErrorString(cudaStatus) << std::endl;
	}
		
	cudaStatus = hipBindTextureToArray(wi_tex_ref, dev_cube_wi_array, &channelFloat);//, hipChannelFormatKindFloat); 	
	hipBindTextureToArray(w_tex_ref, dev_cube_w_array, &channelFloat);//, hipChannelFormatKindFloat);
	
	if (cudaStatus != hipSuccess) {
		std::cout << "error on bind text " << hipGetErrorString(cudaStatus) << std::endl;
	}
	

	const dim3 grid2(((imsize.x + block2.x - 1) / block2.x), ((imsize.y + block2.y - 1) / block2.y));
	
	hipEvent_t start_1, stop_1;
        float time_1;
        hipEventCreate(&start_1);
        hipEventCreate(&stop_1);

        hipEventRecord(start_1);

	slicing <<< grid2, block2 >>> (dev_image, imsize, scale_xy, scale_eps);
	
	hipEventRecord(stop_1);
        hipEventSynchronize(stop_1);

        hipEventElapsedTime(&time_1, start_1, stop_1);


	hipEvent_t start_2, stop_2;
	float time_2;
	hipEventCreate(&start_2);
	hipEventCreate(&stop_2);
	
	hipEventRecord(start_2);
	slicing <<< grid2, block2 >>> (dev_image, imsize, scale_xy, scale_eps);
	hipDeviceSynchronize();
	hipEventRecord(stop_2);
	hipEventSynchronize(stop_2);
	
	hipEventElapsedTime(&time_2, start_2, stop_2);
	
	hipUnbindTexture(wi_tex);
	hipUnbindTexture(w_tex);
	hipFreeArray(dev_cube_wi_array);
	hipFreeArray(dev_cube_w_array);
	float time = time_1 + time_2;
	return time;
}
