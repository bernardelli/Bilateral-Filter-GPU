#include "hip/hip_runtime.h"
#include "slicing.h"

texture<float, 3> wi_tex;
texture<float, 3> w_tex;

__global__ void slicing( float *dev_image, const float *dev_cube_wi, const float *dev_cube_w, const dim3 imsize, int scale_xy, int scale_eps, dim3 dimensions_down)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((i < imsize.x) && (j < imsize.y))
	{

		int k = (int)dev_image[j + imsize.y*i];
		//printf("value = %d, i = %d, j = %d\n", value, i, j);


		//Old try on performing trilinear interpolation
		/*float interpolate_wi0[2][2][2];
		float interpolate_w0[2][2][2];
		
		for( int ii = 0; ii <2; ii++)  {
			int i_idx = floorf(i / scale_xy) + ii;
			for (int jj = 0; jj <2; jj++)  { 
				int j_idx = floorf(j / scale_xy) + jj;
				for (int kk = 0; kk <2; k++)  { 
					int k_idx = floorf(k / scale_eps) + kk;
					int cube_idx = i_idx + dimensions_down.x*j_idx + dimensions_down.x*dimensions_down.y*k_idx;
					interpolate_wi0[ii][jj][kk] = dev_cube_wi[cube_idx];
					interpolate_w0[ii][jj][kk] = dev_cube_w[cube_idx];
				}
			}
		}
		float interpolate_wi1[2][2];
		float interpolate_w1[2][2];
		for( int ii = 0; ii <2; ii++)  {
			for (int jj = 0; jj <2; jj++)  { 
				float k_rest = (k/ scale_eps) - floorf(k/ scale_eps);
				interpolate_wi1[ii][jj] = (1.0-k_rest)*interpolate_wi0[ii][jj][0] + k_rest*interpolate_wi0[ii][jj][1];
				interpolate_w1[ii][jj] = (1.0-k_rest)*interpolate_w0[ii][jj][0] + k_rest*interpolate_w0[ii][jj][1];
			}
		}

		float interpolate_wi2[2];
		float interpolate_w2[2];
		for( int ii = 0; ii <2; ii++)  {
			float j_rest = (j/ scale_xy) - floorf(j/ scale_xy);

			interpolate_wi2[ii] = (1.0-j_rest)*interpolate_wi1[ii][0] + j_rest*interpolate_wi1[ii][1];
			interpolate_w2[ii] = (1.0-j_rest)*interpolate_w1[ii][0] + j_rest*interpolate_w1[ii][1];
			
		}
		float i_rest = (i/ scale_xy) - floorf(i/ scale_xy);*/


		//dev_image[j + imsize.y*i] = ((1.0-i_rest)*interpolate_wi2[0] + i_rest*interpolate_wi2[1])/((1.0-i_rest)*interpolate_w2[0] + i_rest*interpolate_w2[1]);
		dev_image[j + imsize.y*i] = tex3D(wi_tex, 0.5+i/scale_xy, 0.5+j/scale_xy, 0.5+k/scale_eps)/tex3D(w_tex, 0.5+i/scale_xy, 0.5+j/scale_xy, 0.5+k/scale_eps);
		
	}

}

/*__global__ void fill_arrays(hipArray* dev_cube_wi_array, hipArray* dev_cube_w_array, const float* dev_cube_wi, const float* dev_cube_w, const dim3 )
{

	const int ix = blockDim.x*blockIdx.x + threadIdx.x;
	const int iy = blockDim.y*blockIdx.y + threadIdx.y;
	const int iz = blockIdx.z;
	const int cube_idx = ix + iy*dimensions_down.x + iz*dimensions_down.x*dimensions_down.y;
	if(ix < dimensions_down.x &&  iy < dimensions_down.y && iy < dimensions_down.y )
	{
		dev_cube_wi_array = dev_cube_wi[cube_idx];
	}
}*/

int callingSlicing(float* dev_image, const float *dev_cube_wi, const float *dev_cube_w, const dim3 imsize, int scale_xy, int scale_eps, dim3 dimensions_down)
{
	int slicing_status = 0;
	const dim3 block2(16, 16);

	//Calculate grid size to cover the whole image
	const dim3 grid2(((imsize.x + block2.x - 1) / block2.x), ((imsize.y + block2.y - 1) / block2.y));


	hipExtent extent1 = make_hipExtent( dimensions_down.x*sizeof(float), dimensions_down.y, dimensions_down.z); 
	hipExtent extent2 = make_hipExtent( dimensions_down.x, dimensions_down.y, dimensions_down.z); 
	hipArray *dev_cube_wi_array, *dev_cube_w_array;

	hipChannelFormatDesc channelFloat = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&dev_cube_wi_array, &channelFloat, extent2);	
	hipMalloc3DArray(&dev_cube_w_array, &channelFloat, extent2);

	hipMemcpy3DParms copyParams1 = { 0 };
	copyParams1.srcPtr = make_hipPitchedPtr((void*) dev_cube_wi,
		dimensions_down.x * sizeof(float),
		dimensions_down.y, dimensions_down.z);
	copyParams1.dstArray = dev_cube_wi_array;
	copyParams1.extent = extent2;
	copyParams1.kind = hipMemcpyDeviceToDevice;
	hipError_t cudaStatus = hipMemcpy3D(&copyParams1);

	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array1" << std::endl;
		slicing_status = 1;
	}
	hipMemcpy3DParms copyParams2 = { 0 };
	copyParams2.srcPtr = make_hipPitchedPtr((void*) dev_cube_w,
		dimensions_down.x * sizeof(float),
		dimensions_down.y, dimensions_down.z);
	copyParams2.dstArray = dev_cube_w_array;
	copyParams2.extent = extent2;
	copyParams2.kind = hipMemcpyDeviceToDevice;
	cudaStatus = hipMemcpy3D(&copyParams2);
	if (cudaStatus != hipSuccess) {
		std::cout << "error on copying to array2"<< std::endl;
		slicing_status = 1;
	}
	//hipMemcpyToArray(dev_cube_wi_array, 0, 0, dev_cube_wi, dimensions_down.x*dimensions_down.y*dimensions_down.z, hipMemcpyDeviceToDevice);
	//hipMemcpyToArray(dev_cube_w_array, 0, 0, dev_cube_w, dimensions_down.x*dimensions_down.y*dimensions_down.z, hipMemcpyDeviceToDevice);
	//fill_arrays<<grid,block>>(dev_cube_wi_array, dev_cube_w_array,dev_cube_wi, dev_cube_w,dimensions_down);

	//struct hipChannelFormatDesc descr = hipCreateChannelDesc((int)dimensions_down.x, (int)dimensions_down.y, (int)dimensions_down.z, hipChannelFormatKindFloat);
	const textureReference * wi_tex_ref;
	const textureReference * w_tex_ref;
	hipGetTextureReference(&wi_tex_ref, HIP_SYMBOL("wi_tex"));	
	hipGetTextureReference(&w_tex_ref, HIP_SYMBOL("w_tex"));	
 	hipBindTextureToArray(wi_tex_ref, dev_cube_wi_array, &channelFloat);//, hipChannelFormatKindFloat); 	
	hipBindTextureToArray(w_tex_ref, dev_cube_w_array, &channelFloat);//, hipChannelFormatKindFloat);

	
	slicing <<< grid2, block2 >>> (dev_image, dev_cube_wi, dev_cube_w, imsize, scale_xy, scale_eps, dimensions_down);
	hipDeviceSynchronize();

	hipUnbindTexture(wi_tex);
	hipUnbindTexture(w_tex);
	hipFreeArray(dev_cube_wi_array);
	hipFreeArray(dev_cube_w_array);
	return slicing_status;
}
