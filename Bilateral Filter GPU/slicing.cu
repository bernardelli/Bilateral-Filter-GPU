#include "hip/hip_runtime.h"
#include "slicing.h"

__global__ void slicing( float *dev_image, const float *dev_cube_wi, const float *dev_cube_w, const dim3 imsize)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((i < imsize.x) && (j < imsize.y))
	{

		int value = (int)dev_image[j + imsize.y*i];
		//printf("value = %d, i = %d, j = %d\n", value, i, j);

		unsigned int cube_idx = i + imsize.x*j + imsize.x*imsize.y*value;
		dev_image[j + imsize.y*i] = dev_cube_wi[cube_idx] / dev_cube_w[cube_idx];
		//printf("w = %f  wi = %f \n", dev_cube_w[cube_idx], dev_cube_wi[cube_idx]);
		
	}
	//else
		//printf("out of bounds\n");

}

float* callingSlicing( cv::Mat image, float *dev_cube_wi, float *dev_cube_w)
{
	float *dev_image, *result_image; 
	image.convertTo(image, CV_32F);
	int imsize = image.rows*image.cols;
	hipMalloc(&dev_image, imsize*sizeof(float));
	hipMemcpy(dev_image, image.ptr(), imsize*sizeof(float), hipMemcpyHostToDevice);
	
	//Specify a reasonable block size
	const dim3 block2(32, 32);

	//Calculate grid size to cover the whole image
	const dim3 grid2(((image.cols + block2.x - 1) / block2.x), ((image.rows + block2.y - 1) / block2.y));
	
	dim3  image_dimensions = dim3(image.rows, image.cols, 256);
	
	slicing <<< grid2, block2 >>> (dev_image , dev_cube_wi, dev_cube_w, image_dimensions);
	hipDeviceSynchronize();

	result_image = (float*)malloc(imsize*sizeof(float));
	hipMemcpy(result_image, dev_image, imsize*sizeof(float), hipMemcpyDeviceToHost);
}