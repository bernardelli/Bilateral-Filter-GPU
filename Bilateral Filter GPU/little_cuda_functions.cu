#include "little_cuda_functions.h"


/********************************************************************************
*** checking if CUDA is installed and printing out the compute capability and ***
*** the concurrent kernels of each device                                     ***
********************************************************************************/
void checkingDevices()
{
	int deviceCount;
	hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
	
	if (cudaStatus == hipErrorInsufficientDriver) {
		fprintf(stderr, "hipGetDeviceCount failed!  Do you have CUDA installed?");
	}
	
	int device;
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d has compute capability %d.%d and concurrentKernels = %d.\n",
			device, deviceProp.major, deviceProp.minor, deviceProp.concurrentKernels);
	}
	hipDeviceReset();
}


/********************************************************************************
*** getting the size of the malloced space and returning a pointer of that    ***
*** malloced space                                                            ***
********************************************************************************/
float* allocateGpuMemory(int size)
{

	float* p;
	size *= sizeof(float);
	hipError_t err = hipSuccess;
	err = hipMalloc((void **)&p, size);

	if (err != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Error code: %s\n", hipGetErrorString);
		return NULL;
	}

	return p;
}


/********************************************************************************
*** copy memory to the gpu memory                                             ***
********************************************************************************/
hipError_t copyToGpuMem(float *a, float *b, int size)
{
	hipError_t cudaStatus = hipMemcpy(a, b, size * sizeof(float), hipMemcpyHostToDevice);
	return cudaStatus;
}