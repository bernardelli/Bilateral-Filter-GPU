#include "little_cuda_functions.h"

void checkingDevices()
{
	int deviceCount;
	hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
	
	if (cudaStatus == hipErrorInsufficientDriver) {
		fprintf(stderr, "hipGetDeviceCount failed!  Do you have CUDA installed?");
	}
	
	int device;
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d has compute capability %d.%d and concurrentKernels = %d.\n",
			device, deviceProp.major, deviceProp.minor, deviceProp.concurrentKernels);
	}
	hipDeviceReset();
}

hipError_t allocateGpuMemory(char **ptr, int size)
{
	hipError_t cudaStatus = hipMalloc((void**)&ptr, size * sizeof(float));
	return cudaStatus;
}

hipError_t copyToGpuMem(float *a, float *b, int size)
{
	hipError_t cudaStatus = hipMemcpy(a, b, size * sizeof(float), hipMemcpyHostToDevice);
	return cudaStatus;
}