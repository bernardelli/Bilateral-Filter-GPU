#include "hip/hip_runtime.h"
#include "convolution_shared.h"

__global__ void convolution_shared_row(float *output, const float *input, const float *kernel, const int kernel_size, const dim3 imsize)
{	
	const int ix = blockDim.x*blockIdx.x + threadIdx.x;
	const int iy = blockDim.y*blockIdx.y + threadIdx.y;
	const int iz = blockIdx.z;

	
	const int cube_idx = ix + iy*imsize.x + iz*imsize.x*imsize.y;

	const int radius_size = kernel_size / 2;

	extern __shared__ float s_image[]; //size is on kernel call, (block_dim_x + 2 * k_radius_xy)*block_dim_y
	const int s_dim_x = blockDim.x + 2 * radius_size;
	const int s_ix = radius_size + threadIdx.x;
	const int s_iy = threadIdx.y;

	if (threadIdx.x < radius_size) //is on the left part of the shared memory!
	{
		s_image[s_ix - radius_size + s_iy*s_dim_x] = 0.0f;
	}
	if (threadIdx.x >(blockDim.x - radius_size))
	{
		s_image[s_ix + radius_size + s_iy*s_dim_x] = 0.0f;
	}

	s_image[s_ix + s_iy*s_dim_x] = (ix >= imsize.x || iy >= imsize.y || iz >= imsize.z) ? 0.0f : input[cube_idx];
	
	__syncthreads();
	float result = 0.0;

#pragma unroll
	for (int i = 0; i < kernel_size; i++)
	{
		result += kernel[i] * s_image[s_ix - i + radius_size + s_iy*s_dim_x];
	}

	if (ix <imsize.x && iy < imsize.y && iz < imsize.z)
	{

		output[cube_idx] = result;
	}


	//if (result > 0 )
	//	printf("%.01f \n", result);
}


__global__ void convolution_shared_col(float *output, const float *input, const float *kernel, const int kernel_size, const dim3 imsize)
{
	const int ix = blockDim.x*blockIdx.x + threadIdx.x;
	const int iy = blockDim.y*blockIdx.y + threadIdx.y;
	const int iz = blockIdx.z;
	const int cube_idx = ix + iy*imsize.x + iz*imsize.x*imsize.y;

	const int radius_size = kernel_size / 2;

	extern __shared__ float s_image__[]; //size is on kernel call, (block_dim_x + 2 * k_radius_xy)*block_dim_y
	const int s_dim_x = blockDim.x;
	const int s_ix = threadIdx.x;
	const int s_iy = radius_size + threadIdx.y;

	if (threadIdx.y < radius_size) //is on the left part of the shared memory!
	{
		s_image__[s_ix + (s_iy - radius_size)*s_dim_x] = 0.0;
	}
	if (threadIdx.y >(blockDim.y - radius_size))
	{
		s_image__[s_ix + (s_iy + radius_size)*s_dim_x] = 0.0;
	}
	s_image__[s_ix + s_iy*s_dim_x] = (ix >= imsize.x || iy >= imsize.y || iz >= imsize.z) ? 0.0f : input[cube_idx];


	__syncthreads();

	float result = 0.0;

#pragma unroll
	for (int i = 0; i < kernel_size; i++)
	{
		result += kernel[i] * s_image__[s_ix + (s_iy - i + radius_size)*s_dim_x];
	}
	if (ix <imsize.x && iy < imsize.y && iz < imsize.z)
	{

		output[cube_idx] = result;
	}

}


__global__ void convolution_shared_eps(float *output, const float *input, const float *kernel, const int kernel_size, const dim3 imsize)
{
	const int iz = blockDim.x*blockIdx.x + threadIdx.x;
	const int ix = blockDim.y*blockIdx.y + threadIdx.y;
	const int iy = blockIdx.z;
	
	const int cube_idx = ix + iy*imsize.x + iz*imsize.x*imsize.y;

	const int radius_size = kernel_size / 2;

	extern __shared__ float s_image_[]; //size is on kernel call, (block_dim_x + 2 * k_radius_xy)*block_dim_y
	const int s_dim_x = blockDim.x + 2 * radius_size;
	const int s_ix = radius_size + threadIdx.x;
	const int s_iy = threadIdx.y;

	if (threadIdx.x < radius_size) //is on the left part of the shared memory!
	{
		s_image_[s_ix - radius_size + s_iy*s_dim_x] = 0.0;
	}
	if (threadIdx.x >(blockDim.x - radius_size))
	{
		s_image_[s_ix + radius_size + s_iy*s_dim_x] = 0.0;
	}
	s_image_[s_ix + s_iy*s_dim_x] = (ix >= imsize.x || iy >= imsize.y || iz >= imsize.z) ? 0.0f : input[cube_idx];



	__syncthreads();

	float result = 0.0;

#pragma unroll
	for (int i = 0; i < kernel_size; i++)
	{
		result += kernel[i] * s_image_[s_ix - i + radius_size + s_iy*s_dim_x];
	}
	if (ix <imsize.x && iy < imsize.y && iz < imsize.z)
	{

		output[cube_idx] = result;
	}
	

	//if (result > 0 )
		//printf("%.01f \n", result);
}


float callingConvolution_shared(float *dev_cube_wi_out, float *dev_cube_w_out, float *dev_cube_wi, float *dev_cube_w, const float *dev_kernel_xy, int kernel_xy_size, const float *dev_kernel_eps, int kernel_eps_size, dim3  image_dimensions, int device)
{
	/**Getting shared memory size and max block size 
	*/

	//TODO: intitialize this on main
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device); 



	int max_shared_mem = deviceProp.sharedMemPerBlock / sizeof(float);
	int k_radius_xy = kernel_xy_size / 2;
	//Conv row

	//deviceProp.maxThreadsPerMultiProcessor;
	//deviceProp.sharedMemPerMultiprocessor;


	int block_dim_x = image_dimensions.x; //make this work later for big images
	int block_dim_y = max_shared_mem / (block_dim_x + 2 * k_radius_xy);

	if (block_dim_x*block_dim_y > deviceProp.maxThreadsPerBlock)
	{
		block_dim_y = deviceProp.maxThreadsPerBlock / block_dim_x;
	}
	
	int shared_memory_size = sizeof(float)*(block_dim_x + 2 * k_radius_xy)*block_dim_y;


	const dim3 block(block_dim_x, block_dim_y); //threads per block 32 32
	const dim3 grid((image_dimensions.x + block_dim_x - 1) / block_dim_x, (image_dimensions.y + block_dim_y - 1) / block_dim_y, image_dimensions.z);
	
	hipEvent_t start_row_1, stop_row_1, start_row_2, stop_row_2;
	float time_shared_row_1, time_shared_row_2;
	hipEventCreate(&start_row_1);
	hipEventCreate(&stop_row_1);
	hipEventRecord(start_row_1);
	
	convolution_shared_row <<< grid, block, shared_memory_size >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_xy, kernel_xy_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_row_1);
	hipEventSynchronize(stop_row_1);
	hipEventElapsedTime(&time_shared_row_1, start_row_1, stop_row_1);
	
	swap2(&dev_cube_wi_out, &dev_cube_wi);
	
	hipEventCreate(&start_row_2);
	hipEventCreate(&stop_row_2);
	hipEventRecord(start_row_2);
	
	convolution_shared_row <<< grid, block, shared_memory_size >>>(dev_cube_w_out, dev_cube_w, dev_kernel_xy, kernel_xy_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_row_2);
	hipEventSynchronize(stop_row_2);
	hipEventElapsedTime(&time_shared_row_2, start_row_2, stop_row_2);
	
	swap2(&dev_cube_w_out, &dev_cube_w);

	//Conv Col



	block_dim_y = image_dimensions.y; //make this work later for big images
	block_dim_x = max_shared_mem / (block_dim_y + 2 * k_radius_xy);

	if (block_dim_x*block_dim_y > deviceProp.maxThreadsPerBlock)
	{
		block_dim_x = deviceProp.maxThreadsPerBlock / block_dim_y;
	}

	shared_memory_size = sizeof(float)*block_dim_x*(block_dim_y + 2 * k_radius_xy);


	const dim3 block2(block_dim_x, block_dim_y); //threads per block 32 32
	const dim3 grid2((image_dimensions.x + block_dim_x - 1) / block_dim_x, (image_dimensions.y + block_dim_y - 1) / block_dim_y, image_dimensions.z);
	
	hipEvent_t start_col_1, stop_col_1, start_col_2, stop_col_2;
	float time_shared_col_1, time_shared_col_2;
	hipEventCreate(&start_col_1);
	hipEventCreate(&stop_col_1);
	hipEventRecord(start_col_1);
	
	convolution_shared_col <<< grid2, block2, shared_memory_size >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_xy, kernel_xy_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_col_1);
	hipEventSynchronize(stop_col_1);
	hipEventElapsedTime(&time_shared_col_1, start_col_1, stop_col_1);
	
	swap2(&dev_cube_wi_out, &dev_cube_wi);
	
	hipEventCreate(&start_col_2);
	hipEventCreate(&stop_col_2);
	hipEventRecord(start_col_2);
	
	convolution_shared_col <<< grid2, block2, shared_memory_size >>>(dev_cube_w_out, dev_cube_w, dev_kernel_xy, kernel_xy_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_col_2);
	hipEventSynchronize(stop_col_2);
	hipEventElapsedTime(&time_shared_col_2, start_col_2, stop_col_2);
	
	swap2(&dev_cube_w_out, &dev_cube_w);
	// conv eps

	//Conv Col

	int k_radius_eps = kernel_eps_size / 2;

	int block_dim_eps = image_dimensions.z; //make this work later for big images
	block_dim_x = max_shared_mem / (block_dim_eps + 2 * k_radius_eps);

	if (block_dim_eps*block_dim_x > deviceProp.maxThreadsPerBlock)
	{
		block_dim_x = deviceProp.maxThreadsPerBlock / block_dim_eps;
	}

	shared_memory_size = sizeof(float)*(block_dim_eps + 2 * k_radius_eps)*(block_dim_x);


	const dim3 block3(block_dim_eps, block_dim_x); //threads per block 32 32
	const dim3 grid3((image_dimensions.z + block_dim_eps - 1) / block_dim_eps, (image_dimensions.x + block_dim_x - 1) / block_dim_x, image_dimensions.y);

	hipEvent_t start_eps_1, stop_eps_1, start_eps_2, stop_eps_2;
	float time_shared_eps_1, time_shared_eps_2;
	hipEventCreate(&start_eps_1);
	hipEventCreate(&stop_eps_1);
	hipEventRecord(start_eps_1);
	
	convolution_shared_eps <<< grid3, block3, shared_memory_size >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_eps, kernel_eps_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_eps_1);
	hipEventSynchronize(stop_eps_1);
	hipEventElapsedTime(&time_shared_eps_1, start_eps_1, stop_eps_1);
	
	hipEventCreate(&start_eps_2);
	hipEventCreate(&stop_eps_2);
	hipEventRecord(start_eps_2);
	
	convolution_shared_eps <<< grid3, block3, shared_memory_size >>>(dev_cube_w_out, dev_cube_w, dev_kernel_eps, kernel_eps_size, image_dimensions);
	hipDeviceSynchronize();
	
	hipEventRecord(stop_eps_2);
	hipEventSynchronize(stop_eps_2);
	hipEventElapsedTime(&time_shared_eps_2, start_eps_2, stop_eps_2);
	float time = time_shared_row_1 + time_shared_row_2 + time_shared_col_1 + time_shared_col_2 + time_shared_eps_1 + time_shared_eps_2;
	return time;
}

void swap2(float** a, float** b){
	float* c = *a;
    *a = *b;
    *b = c;
}