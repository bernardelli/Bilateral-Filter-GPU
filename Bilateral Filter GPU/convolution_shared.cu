#include "hip/hip_runtime.h"
#include "convolution_shared.h"

__global__ void convolution__shared_row(float *output, const float *input, const float *kernel, const int ksize, const dim3 imsize)
{	
	unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;
	unsigned int i = ix + iy*blockDim.x*gridDim.x;
	//unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int imsize_x = imsize.x;
	unsigned int imsize_y = imsize.y;
	unsigned int imsize_z = imsize.z;
	unsigned int im_size = imsize_x*imsize_y*imsize_z;
	//printf("i = %d\n", i);
	//idx = x_i + imsize_x*y_i + imsize_x*imsize_y*z_i
	unsigned int z_i = i / (unsigned int) (imsize_x*imsize_y);
	unsigned int y_i = (i - z_i*(unsigned int)(imsize_x*imsize_y)) / imsize_x;
	unsigned int x_i = i - y_i*imsize_x - z_i*(unsigned int)(imsize_x*imsize_y);

	__shared__ float s_image[BLOCK_DIM][BLOCK_DIM];
	
	/*
	double result = 0.0;
	unsigned int idx = 0;
	unsigned int k_offset = (ksize / 2);
	for (int k = 0; k < ksize; k++) {
		if (dir == X_DIR) {
			int x_input = k_offset - k + x_i;
			if (x_input >= 0 && x_input < imsize_x) {
				idx = (unsigned int)x_input + imsize_x*y_i + imsize_x*imsize_y*z_i;
				if (idx < im_size)
					result += input[idx]*kernel[k];
			}
		}
		else if (dir == Y_DIR) {
			int y_input = k_offset - k + y_i;

			if (y_input >= 0 && y_input < imsize_y) {
				idx = x_i + imsize_x*(unsigned int)y_input + imsize_x*imsize_y*z_i;
				if (idx < im_size)
					result += input[idx] * kernel[k];
			}
			//else
				//printf("out of bounds\n");
		}
		else if (dir == Z_DIR) {
			int z_input = k_offset - k + z_i;
			//printf("z_input %f\n", z_input);
			if (z_input >= 0 && z_input < imsize_z) {
				idx = x_i + imsize_x*y_i + imsize_x*imsize_y*(unsigned int) z_input;
				if (idx < im_size)
					result += input[idx] * kernel[k];
			}
			//else
				//printf("out of bounds\n");
		}
		else
			printf("All wrong");
		
	}
	idx = x_i + imsize_x*y_i + imsize_x*imsize_y*z_i;
	if (idx < im_size)
		output[idx] = result;
		*/
}

void callingConvolution_shared(float *dev_cube_wi_out, float *dev_cube_w_out, float *dev_cube_wi, float *dev_cube_w, const float *dev_kernel_xy, int kernel_xy_size, const float *dev_kernel_eps, int kernel_eps_size, dim3  image_dimensions)
{
	/**Getting shared memory size and max block size 
	*/
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // device = 0;

	int max_shared_mem = deviceProp.sharedMemPerBlock/sizeof(float);
	
	//deviceProp.maxThreadsPerMultiProcessor;
	//deviceProp.sharedMemPerMultiprocessor;
	int k_radius_xy = kernel_xy_size / 2;
	int regular_block_x_dim = 32;
	int block_dim_x = (k_radius_xy > regular_block_x_dim) ? k_radius_xy : regular_block_x_dim;
	int block_dim_y = max_shared_mem / (block_dim_x + 2 * k_radius_xy);

	if (block_dim_x*block_dim_y > deviceProp.maxThreadsPerBlock)
	{
		block_dim_y = block_dim_x / deviceProp.maxThreadsPerBlock;
	}



	const dim3 block(BLOCK_DIM, BLOCK_DIM); //threads per block 32 32

	int grin = 256;
	const dim3 grid(grin, grin);
	/*
	
	convolution <<< grid, block >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_xy, kernel_xy_size, image_dimensions, X_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_wi, &dev_cube_wi_out);

	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel_xy, kernel_xy_size, image_dimensions, X_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_w, &dev_cube_w_out);
		
	convolution <<< grid, block >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_xy, kernel_xy_size, image_dimensions, Y_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_wi, &dev_cube_wi_out);
	
	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel_xy, kernel_xy_size, image_dimensions, Y_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_w, &dev_cube_w_out);
	
	convolution <<< grid, block >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel_eps, kernel_eps_size, image_dimensions, Z_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_wi, &dev_cube_wi_out);
	
	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel_eps, kernel_eps_size, image_dimensions, Z_DIR);
	hipDeviceSynchronize();
	swap(&dev_cube_w, &dev_cube_w_out);*/
	
}
/*
void swap(float** a, float** b){
	float* c = *a;
    *a = *b;
    *b = c;
	}*/