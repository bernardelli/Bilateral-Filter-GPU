#include "hip/hip_runtime.h"
#include "convolution.h"

__global__ void convolution(float *output, const float *input, const float *kernel, const int ksize, const dim3 imsize, const int dir)
{	
	unsigned int ix = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;
	unsigned int i = ix + iy*blockDim.x*gridDim.x;
	//unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int imsize_x = imsize.x;
	unsigned int imsize_y = imsize.y;
	unsigned int imsize_z = imsize.z;
	unsigned int im_size = imsize_x*imsize_y*imsize_z;
	//printf("i = %d\n", i);
	//idx = x_i + imsize_x*y_i + imsize_x*imsize_y*z_i
	unsigned int z_i = i / (unsigned int) (imsize_x*imsize_y);
	unsigned int y_i = (i - z_i*(unsigned int)(imsize_x*imsize_y)) / imsize_x;
	unsigned int x_i = i - y_i*imsize_x - z_i*(unsigned int)(imsize_x*imsize_y);
	
	
	double result = 0.0;
	unsigned int idx = 0;
	unsigned int k_offset = (ksize / 2);
	for (int k = 0; k < ksize; k++) {
		if (dir == X_DIR) {
			int x_input = k_offset - k + x_i;
			if (x_input >= 0 && x_input < imsize_x) {
				idx = (unsigned int)x_input + imsize_x*y_i + imsize_x*imsize_y*z_i;
				if (idx < im_size)
					result += input[idx]*kernel[k];
			}
		}
		else if (dir == Y_DIR) {
			int y_input = k_offset - k + y_i;

			if (y_input >= 0 && y_input < imsize_y) {
				idx = x_i + imsize_x*(unsigned int)y_input + imsize_x*imsize_y*z_i;
				if (idx < im_size)
					result += input[idx] * kernel[k];
			}
			//else
				//printf("out of bounds\n");
		}
		else if (dir == Z_DIR) {
			int z_input = k_offset - k + z_i;
			//printf("z_input %f\n", z_input);
			if (z_input >= 0 && z_input < imsize_z) {
				idx = x_i + imsize_x*y_i + imsize_x*imsize_y*(unsigned int) z_input;
				if (idx < im_size)
					result += input[idx] * kernel[k];
			}
			//else
				//printf("out of bounds\n");
		}
		else
			printf("All wrong");
		
	}
	idx = x_i + imsize_x*y_i + imsize_x*imsize_y*z_i;
	if (idx < im_size)
		output[idx] = result;
}

void callingConvolution(cv::Mat image, float *dev_cube_wi_out, float *dev_cube_w_out, float *dev_cube_wi, float *dev_cube_w, float *dev_kernel, int kernel_size)
{
	const dim3 block(32, 32); //threads per block 32 32

	int grin = 256;
	const dim3 grid(grin, grin);
	
	dim3  image_dimensions = dim3(image.rows, image.cols, 256);
	
	convolution <<< grid, block >>>
		(dev_cube_wi_out, 
		dev_cube_wi, 
		dev_kernel, 
		kernel_size, 
		image_dimensions, 
		X_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_wi, dev_cube_wi_out);

	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel, kernel_size, image_dimensions, X_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_w, dev_cube_w_out);
		
	convolution <<< grid, block >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel, kernel_size, image_dimensions, Y_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_wi, dev_cube_wi_out);
	
	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel, kernel_size, image_dimensions, Y_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_w, dev_cube_w_out);
	
	convolution <<< grid, block >>>(dev_cube_wi_out, dev_cube_wi, dev_kernel, kernel_size, image_dimensions, Z_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_wi, dev_cube_wi_out);
	
	convolution <<< grid, block >>>(dev_cube_w_out, dev_cube_w, dev_kernel, kernel_size, image_dimensions, Z_DIR);
	hipDeviceSynchronize();
	swap(dev_cube_w, dev_cube_w_out);
	
}

void swap(float*& a, float*& b){
    float* c = a;
    a = b;
    b = c;
}