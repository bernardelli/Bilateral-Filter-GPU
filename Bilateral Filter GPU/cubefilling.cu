#include "hip/hip_runtime.h"
#include "cubefilling.cuh"


/*
__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x ;
	c[index] = a[index] + b[index];
}
*/

__global__ void cubefilling(const float* image, float *dev_cube_wi, float *dev_cube_w, const dim3 image_size, int scale_xy, int scale_eps, dim3 dimensions_down)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < image_size.x && j < image_size.y) {
		
		unsigned int k = (unsigned int)image[j + image_size.y*i];
		unsigned int cube_idx = floorf(i/scale_xy) + dimensions_down.x*floorf(j/scale_xy) + dimensions_down.x*dimensions_down.y*floorf(k/scale_eps);
		

		atomicAdd(&dev_cube_wi[cube_idx], (float) k);
		atomicAdd(&dev_cube_w[cube_idx], 1.0);
		//dev_cube_wi[cube_idx] += ((float)k);

		//dev_cube_w[cube_idx] += 1.0;

		//Next level: perform filling and Z convolution at the same time!
	}


}

float callingCubefilling(const float* dev_image, float *dev_cube_wi, float *dev_cube_w, const dim3 image_size, int scale_xy, int scale_eps, dim3 dimensions_down)
{

	
	dim3 dimBlock(16, 16);
	dim3 dimGrid((image_size.x + dimBlock.x - 1) / dimBlock.x,
		(image_size.y + dimBlock.y - 1) / dimBlock.y);

	//hipMemset(dev_cube_wi, 0, image_size.x*image_size.y*image_size.z*sizeof(float)); //seems to be useless
	//hipMemset(dev_cube_w, 0, image_size.x*image_size.y*image_size.z*sizeof(float));
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	
	cubefilling <<< dimGrid, dimBlock >>>(dev_image, dev_cube_wi, dev_cube_w, image_size, scale_xy, scale_eps, dimensions_down);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time, start, stop);
	
	return time;

}

